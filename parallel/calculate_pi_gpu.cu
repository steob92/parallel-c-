
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <omp.h>

// Function to calculate pi
__global__
void assemble_pi (int num_steps, float *pi){

    float step = 1.0/(float) num_steps;
    float x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < num_steps; i += stride){
        x = (i+0.5)*step;
        // Sum over each's own sum counter
        // sum[i] = 4.0/(1.0+x*x) * step;
        atomicAdd(&pi[0], 4.0/(1.0+x*x) * step);
    }
}

float calculate_pi (int num_steps){

    float *_pi;
    float pi;
    float step = 1.0/(float) num_steps;

    int blockSize = 1<<10;
    int numBlocks = (num_steps + blockSize - 1) / blockSize;
    // cudaMallocManaged(&sum, num_steps*sizeof(float));
    hipMallocManaged(&_pi, sizeof(float));
    _pi[0] = 0;
    assemble_pi<<<numBlocks, blockSize>>>(num_steps, _pi);

    hipDeviceSynchronize();
    pi = _pi[0];
    hipFree(_pi);

    return pi;
}



int main(){

    // 1 million steps
    int num_steps = 1000000;
    int n_repeat = 500;

    // mean pi value
    float serial_pi = 0;

    // Run timer
    auto start_time = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < n_repeat ; i ++) serial_pi += calculate_pi(num_steps);
    auto end_time = std::chrono::high_resolution_clock::now();
    
    std::chrono::duration<float> serial_duration = end_time - start_time;

    // Print average results
    std::cout << "Serial Calculation of Pi: " << serial_pi / n_repeat 
              << std::endl << "Duration: " << serial_duration.count() / n_repeat<< " seconds" << std::endl;

}